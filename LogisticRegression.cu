#include "hip/hip_runtime.h"
#include "Helper.h"
#include "ArffImporter.h"

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>


#define WARP_SIZE 32

Node initNode( unsigned int numFeatures )
{
    Node node;
    node.numFeatures = numFeatures;
    node.weights = (float*) malloc( (numFeatures + 1) * sizeof( float ) );
    memset( node.weights, 0, (numFeatures + 1) * sizeof( float ) );

    return node;
}

void normalize(
    std::vector<NumericAttr> featureVec,
    float* featureMat,
    float* featureMatTrans,
    unsigned int numInstances )
{
    unsigned int numFeatures = featureVec.size();

    for (unsigned int i = 0; i < numFeatures; i++)
    {
        // Use either range / standard deviation
        float range = featureVec[i].max - featureVec[i].min;
        if (range == 0.0) continue;

        for (unsigned int j = 0; j < numInstances; j++)
        {
            unsigned int featureIndex = j * numFeatures + i;
            featureMat[featureIndex] =
                (featureMat[featureIndex] - featureVec[i].mean) / range;
            featureMatTrans[i * numInstances + j] = featureMat[featureIndex];
        }
    }
}

__device__ __forceinline__ float shuffleSum32( float sum )
{
    // Reduce final warp using shuffle
    for (unsigned short shift = WARP_SIZE / 2; shift > 0; shift >>= 1)
        sum += __shfl_down( sum, shift );

    return sum;
}

// Parallel sum combining shuffle and shared memory
__device__ __forceinline__ float parallelSum512(
    float* __restrict__ sharedData )
{
    float sum = sharedData[threadIdx.x];

    if (threadIdx.x < 256)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 256];
    __syncthreads();

    if (threadIdx.x < 128)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 128];
    __syncthreads();

    if (threadIdx.x < 64)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 64];
    __syncthreads();

#if (__CUDA_ARCH__ >= 300)
    if (threadIdx.x < 32)
    {
        sum += sharedData[threadIdx.x + 32];
        // Reduce final warp using shuffle
        sum = shuffleSum32( sum );
    }
#else
    // fully unroll reduction within a single warp
    if (threadIdx.x < 32)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 32];
    __syncthreads();

    if (threadIdx.x < 16)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 16];
    __syncthreads();

    if (threadIdx.x < 8)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 8];
    __syncthreads();

    if (threadIdx.x < 4)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 4];
    __syncthreads();

    if (threadIdx.x < 2)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 2];
    __syncthreads();

    if (threadIdx.x < 1)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 1];
    __syncthreads();
#endif

    return sum;
}

// Parallel sum combining shuffle and shared memory
__device__ __forceinline__ float parallelSum256(
    float* __restrict__ sharedData )
{
    float sum = sharedData[threadIdx.x];

    if (threadIdx.x < 128)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 128];
    __syncthreads();

    if (threadIdx.x < 64)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 64];
    __syncthreads();

#if (__CUDA_ARCH__ >= 300)
    if (threadIdx.x < 32)
    {
        sum += sharedData[threadIdx.x + 32];
        // Reduce final warp using shuffle
        sum = shuffleSum32( sum );
    }
#else
    // fully unroll reduction within a single warp
    if (threadIdx.x < 32)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 32];
    __syncthreads();

    if (threadIdx.x < 16)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 16];
    __syncthreads();

    if (threadIdx.x < 8)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 8];
    __syncthreads();

    if (threadIdx.x < 4)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 4];
    __syncthreads();

    if (threadIdx.x < 2)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 2];
    __syncthreads();

    if (threadIdx.x < 1)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 1];
    __syncthreads();
#endif

    return sum;
}

// Parallel sum combining shuffle and shared memory
__device__ __forceinline__ float parallelSum128(
    float* __restrict__ sharedData )
{
    float sum = sharedData[threadIdx.x];

    if (threadIdx.x < 64)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 64];
    __syncthreads();

#if (__CUDA_ARCH__ >= 300)
    if (threadIdx.x < 32)
    {
        sum += sharedData[threadIdx.x + 32];
        // Reduce final warp using shuffle
        sum = shuffleSum32( sum );
    }
#else
    // fully unroll reduction within a single warp
    if (threadIdx.x < 32)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 32];
    __syncthreads();

    if (threadIdx.x < 16)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 16];
    __syncthreads();

    if (threadIdx.x < 8)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 8];
    __syncthreads();

    if (threadIdx.x < 4)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 4];
    __syncthreads();

    if (threadIdx.x < 2)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 2];
    __syncthreads();

    if (threadIdx.x < 1)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 1];
    __syncthreads();
#endif

    return sum;
}

// Parallel sum combining shuffle and shared memory
__device__ __forceinline__ float parallelSum64(
    float* __restrict__ sharedData )
{
    float sum = sharedData[threadIdx.x];

#if (__CUDA_ARCH__ >= 300)
    if (threadIdx.x < 32)
    {
        sum += sharedData[threadIdx.x + 32];
        // Reduce final warp using shuffle
        sum = shuffleSum32( sum );
    }
#else
    // fully unroll reduction within a single warp
    if (threadIdx.x < 32)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 32];
    __syncthreads();

    if (threadIdx.x < 16)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 16];
    __syncthreads();

    if (threadIdx.x < 8)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 8];
    __syncthreads();

    if (threadIdx.x < 4)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 4];
    __syncthreads();

    if (threadIdx.x < 2)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 2];
    __syncthreads();

    if (threadIdx.x < 1)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 1];
    __syncthreads();
#endif

    return sum;
}

__global__ void ComputeCost(
    float* __restrict__ dCostArr,
    const unsigned short* __restrict__ dClassArr,
    const unsigned int numInstances )
{
    unsigned int instanceId = blockIdx.x * blockDim.x + threadIdx.x;
    if (instanceId >= numInstances) return;

    float cost = dCostArr[instanceId];
    cost = 1.0 / (1.0 + exp(-cost)) - (float) dClassArr[instanceId];
    dCostArr[instanceId] = cost;
}

__global__ void VecDotMat(
    float* __restrict__ dResVec,
    const float* __restrict__ dVec,
    const float* __restrict__ dMat,
    const unsigned int numColumns,
    const unsigned int numRows )
{
    unsigned int rowId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int columnId = threadIdx.x * 2;
    if (rowId >= numRows) return;

    const float* __restrict__ dRow = dMat + rowId * numColumns;
    float sum = 0.0f;
    if (numColumns < 32 && blockDim.x == 32)
    {
        if (threadIdx.x < numColumns)
            sum = dRow[threadIdx.x] * dVec[threadIdx.x];
        sum = shuffleSum32( sum );
    }
    else
    {
        extern __shared__ float sharedPartSum[];
        if (columnId < numColumns)
            sum += dRow[columnId] * dVec[columnId];
        if (columnId + 1 < numColumns)
            sum += dRow[columnId + 1] * dVec[columnId + 1];
        sharedPartSum[threadIdx.x] = sum;
        __syncthreads();

        switch (blockDim.x)
        {
            case 64:
                sum = parallelSum64( sharedPartSum );
                break;
            case 128:
                sum = parallelSum128( sharedPartSum );
                break;
            case 256:
                sum = parallelSum256( sharedPartSum );
                break;
            case 512:
                sum = parallelSum512( sharedPartSum );
                break;
            default:
                break;
        }
    }

    if (threadIdx.x == 0) dResVec[rowId] = sum;
}

__global__ void VecDotMatPartial(
    float* __restrict__ dPartSumArr,
    const float* __restrict__ dVec,
    const float* __restrict__ dMat,
    const unsigned int partSumLen,
    const unsigned int numColumns,
    const unsigned int numRows )
{
    unsigned int rowId = blockIdx.y;
    unsigned int columnId = blockIdx.x * blockDim.x * 2 + threadIdx.x * 2;
    if (rowId >= numRows) return;

    const float* __restrict__ dRow = dMat + rowId * numColumns;
    float sum = 0.0;

    extern __shared__ float sharedPartSum[];
    if (columnId < numColumns)
        sum += dRow[columnId] * dVec[columnId];
    if (columnId + 1 < numColumns)
        sum += dRow[columnId + 1] * dVec[columnId + 1];
    sharedPartSum[threadIdx.x] = sum;
    __syncthreads();

    sum = parallelSum512( sharedPartSum );
    if (threadIdx.x == 0)
        dPartSumArr[partSumLen * rowId + blockIdx.x] = sum;
}

// Reduce partial sum
__global__ void SumColumns(
    float* __restrict__ dResVec,
    const float* __restrict__ dMat,
    const unsigned int numColumns,
    const unsigned int numRows )
{
    if (blockIdx.x >= numRows) return;

    float sum = 0.0f;
    if (blockDim.x == 32)
    {
        if (threadIdx.x < numColumns)
            sum = dMat[blockIdx.x * numColumns + threadIdx.x];
        sum = shuffleSum32( sum );
    }
    else
    {
        extern __shared__ float sharedPartSum[];
        unsigned int columnId = threadIdx.x * 2;
        if (columnId < numColumns)
            sum += dMat[columnId];
        if (columnId + 1 < numColumns)
            sum += dMat[columnId + 1];
        sharedPartSum[threadIdx.x] = sum;
        __syncthreads();

        switch (blockDim.x)
        {
            case 64:
                sum = parallelSum64( sharedPartSum );
                break;
            case 128:
                sum = parallelSum128( sharedPartSum );
                break;
            case 256:
                sum = parallelSum256( sharedPartSum );
                break;
            case 512:
                sum = parallelSum512( sharedPartSum );
                break;
            default:
                break;
        }
    }

    if (threadIdx.x == 0) dResVec[blockIdx.x] = sum;
}

__global__ void UpdateWeight(
    float* __restrict__ dWeightArr,
    const float* __restrict__ dSumArr,
    const unsigned int alpha,
    const unsigned int numInstances,
    const unsigned int numFeatures )
{
    unsigned int featureId = blockIdx.x * blockDim.x + threadIdx.x;
    if (featureId >= numFeatures) return;

    dWeightArr[featureId] -=
        alpha / (float) numInstances * dSumArr[featureId];

    if (featureId == 0)
        printf( "Updating weights completed, weight: %f\n", dWeightArr[0] );
}

inline void cudaErrorCheck( hipError_t cudaRes )
{
    if (cudaRes != hipSuccess)
        printf(
            "kernel launch failed with error \"%s\".\n",
            hipGetErrorString( cudaRes )
        );
}

int main()
{
    ArffImporter trainSetImporter;
    trainSetImporter.Read( "Dataset/train/train-first1000.arff" );

    // ArffImporter testSetImporter;
    // testSetImporter.Read( "Dataset/test/dev-first1000.arff" );

    unsigned int numInstances = trainSetImporter.GetNumInstances();
    float* featureMat = trainSetImporter.GetFeatureMat();
    float* featureMatTrans = trainSetImporter.GetFeatureMatTrans();
    unsigned short* classArr = trainSetImporter.GetClassIndex();
    std::vector<NumericAttr> featureVec = trainSetImporter.GetFeatures();
    unsigned int numFeatures = featureVec.size();

    normalize( featureVec, featureMat, featureMatTrans, numInstances );
    Node node = initNode( numFeatures );

    /*-- Determine block and grid size of Weight Dot FeatureMat kernel --*/
    // dim3 preBlockDimL1;
    // dim3 preGridDimL1;
    // dim3 preBlockDimL2;
    // dim3 preGridDimL2;
    // unsigned int prePartSumLen;
    // unsigned int preSharedMemoSizeL1;
    // unsigned int preSharedMemoSizeL2;
    // // Assume numFeatures < 1024
    // if (numFeatures > 1024)
    // {
    //     preBlockDimL1.x = 512;
    //     uwGridDimL1.x = (numFeatures + 1023) / 1024;
    //     uwGridDimL1.y = numInstances;
    //     prePartSumLen = uwGridDimL1.x;
    //     sharedMemoSizeL1 = 512 * sizeof( float );

    //     uwGridDimL2.x = numInstances;
    //     // Assume uwPartSumLen <= 1024
    //     if (uwPartSumLen <= 32)
    //     {
    //         uwBlockDimL2.x = 32;
    //         sharedMemoSizeL2 = 0;
    //     }
    //     else
    //     {
    //         if (prePartSumLen <= 64) uwBlockDimL2.x = 32;
    //         else if (prePartSumLen <= 128) uwBlockDimL2.x = 64;
    //         else if (prePartSumLen <= 256) uwBlockDimL2.x = 128;
    //         else if (prePartSumLen <= 512) uwBlockDimL2.x = 256;
    //         else uwBlockDimL2.x = 512;
    //         sharedMemoSizeL2 = uwBlockDimL2.x * sizeof( float );
    //     }
    // }
    // else
    // {
    //     uwPartSumLen = 1;
    //     uwGridDimL1.x = numFeatures;
    //     if (numFeatures <= 32)
    //     {
    //         preBlockDimL1.x = 32;
    //         sharedMemoSizeL1 = 0;
    //     }
    //     else
    //     {
    //         if (numFeatures <= 64) preBlockDimL1.x = 32;
    //         else if (numFeatures <= 128) preBlockDimL1.x = 64;
    //         else if (numFeatures <= 256) preBlockDimL1.x = 128;
    //         else if (numFeatures <= 512) preBlockDimL1.x = 256;
    //         sharedMemoSizeL1 = preBlockDimL1.x * sizeof( float );
    //     }
    // }

    dim3 dotBlockDim;
    dim3 dotGridDim;
    unsigned int preSharedMemoSizeL1 = 512 * sizeof( float );
    unsigned int prePartSumLen = 1;
    // Assume numFeatures <= 1024 (max number of threads per block)
    // dotBlockDim.x = numFeatures;
    dotBlockDim.x = 512;
    if (numInstances < 1024)
        dotGridDim.x = numInstances;
    else
    {
        dotGridDim.x = 1024;
        dotGridDim.y = (numInstances + dotGridDim.x - 1) / dotGridDim.x;
    }

    /* Determine block and grid size of ComputeCost & UpdateWeight kernel */
    dim3 uwBlockDim;
    dim3 uwGridDim;
    if (numFeatures > 1024)
    {
        uwBlockDim.x = 1024;
        uwGridDim.x = (numInstances + 1023) / 1024;
    }
    else uwBlockDim.x = numFeatures;

    /* Determine block and grid size of ComputeCost & UpdateWeight kernel */
    dim3 ccBlockDim;
    dim3 ccGridDim;
    if (numInstances > 1024)
    {
        ccBlockDim.x = 1024;
        ccGridDim.x = (numInstances + 1023) / 1024;
    }
    else ccBlockDim.x = numInstances;

    /*- Determine block and grid size of Cost Dot FeatureMatTrans kernel -*/
    dim3 uwBlockDimL1;
    dim3 uwGridDimL1;
    dim3 uwBlockDimL2;
    dim3 uwGridDimL2;
    unsigned int uwPartSumLen;
    unsigned int uwSharedMemoSizeL1;
    unsigned int uwSharedMemoSizeL2;
    // Assume numFeatures < 1024
    if (numInstances > 1024)
    {
        uwBlockDimL1.x = 512;
        uwGridDimL1.x = (numInstances + 1023) / 1024;
        uwGridDimL1.y = numFeatures;
        uwPartSumLen = uwGridDimL1.x;
        uwSharedMemoSizeL1 = 512 * sizeof( float );

        uwGridDimL2.x = numFeatures;
        // Assume uwPartSumLen <= 1024
        if (uwPartSumLen <= 32)
        {
            uwBlockDimL2.x = 32;
            uwSharedMemoSizeL2 = 0;
        }
        else
        {
            if (uwPartSumLen <= 64) uwBlockDimL2.x = 32;
            else if (uwPartSumLen <= 128) uwBlockDimL2.x = 64;
            else if (uwPartSumLen <= 256) uwBlockDimL2.x = 128;
            else if (uwPartSumLen <= 512) uwBlockDimL2.x = 256;
            else uwBlockDimL2.x = 512;
            uwSharedMemoSizeL2 = uwBlockDimL2.x * sizeof( float );
        }
    }
    else
    {
        uwPartSumLen = 1;
        uwGridDimL1.x = numFeatures;
        if (numInstances <= 32)
        {
            uwBlockDimL1.x = 32;
            uwSharedMemoSizeL1 = 0;
        }
        else
        {
            if (numInstances <= 64) uwBlockDimL1.x = 32;
            else if (numInstances <= 128) uwBlockDimL1.x = 64;
            else if (numInstances <= 256) uwBlockDimL1.x = 128;
            else if (numInstances <= 512) uwBlockDimL1.x = 256;
            uwSharedMemoSizeL1 = uwBlockDimL1.x * sizeof( float );
        }
    }

    float* dCostArr;
    float* dWeightArr;
    float* dFeatureMat;
    float* dFeatureMatTrans;
    float* dPrePartSumArr;
    float* dUWPartSumArr;
    float* dUWSumArr;
    unsigned short* dClassArr;
    cudaErrorCheck( hipMalloc( (void**) &dWeightArr, (numFeatures + 1) * sizeof( float ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dCostArr, numInstances * sizeof( float ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dFeatureMat, numInstances * numFeatures * sizeof( float ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dFeatureMatTrans, numInstances * numFeatures * sizeof( float ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dClassArr, numInstances * sizeof( unsigned short ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dUWSumArr, numFeatures * sizeof( float ) ) );
    if (uwPartSumLen > 1)
        cudaErrorCheck( hipMalloc( (void**) &dUWPartSumArr, uwPartSumLen * numFeatures * sizeof( float ) ) );
    if (prePartSumLen > 1)
        cudaErrorCheck( hipMalloc( (void**) &dPrePartSumArr, prePartSumLen * numInstances * sizeof( float ) ) );
    cudaErrorCheck( hipMemcpyAsync(
        dFeatureMat,
        featureMat,
        numInstances * numFeatures * sizeof( float ),
        hipMemcpyHostToDevice ) );
    cudaErrorCheck( hipMemcpyAsync(
        dFeatureMatTrans,
        featureMatTrans,
        numInstances * numFeatures * sizeof( float ),
        hipMemcpyHostToDevice ) );
    cudaErrorCheck( hipMemcpyAsync(
        dWeightArr,
        node.weights,
        (numFeatures + 1) * sizeof( float ),
        hipMemcpyHostToDevice ) );
    cudaErrorCheck( hipMemcpyAsync(
        dClassArr,
        classArr,
        numInstances * sizeof( unsigned short ),
        hipMemcpyHostToDevice ) );

    // Gradient descent params
    unsigned int alpha = 50;
    unsigned int maxIter = 200;
    unsigned int iter = 0;

    time_t start, end;
    float dif;
    time( &start );
    
    printf( "\nStart gradient descent...\n" );

    // Gradient descent
    while (iter++ < maxIter)
    {
        // Predict
        VecDotMat<<< dotGridDim, dotBlockDim, preSharedMemoSizeL1 >>>(
            dCostArr,
            dWeightArr,
            dFeatureMat,
            numFeatures,
            numInstances );
        cudaErrorCheck( hipGetLastError() );

        ComputeCost<<< ccGridDim, ccBlockDim >>>(
            dCostArr,
            dClassArr,
            numInstances );
        cudaErrorCheck( hipGetLastError() );

        // Update weights
        if (uwPartSumLen == 1)
        {
            VecDotMat<<< uwGridDimL1, uwBlockDimL1, uwSharedMemoSizeL1 >>>(
                dUWSumArr,
                dCostArr,
                dFeatureMatTrans,
                numInstances,
                numFeatures );
            cudaErrorCheck( hipGetLastError() );
        }
        else
        {
            VecDotMatPartial<<< uwGridDimL1, uwBlockDimL1, uwSharedMemoSizeL1 >>>(
                dUWPartSumArr,
                dCostArr,
                dFeatureMatTrans,
                uwPartSumLen,
                numInstances,
                numFeatures );
            cudaErrorCheck( hipGetLastError() );
            SumColumns<<< uwGridDimL2, uwBlockDimL2, uwSharedMemoSizeL2 >>>(
                dUWSumArr,
                dUWPartSumArr,
                uwPartSumLen,
                numFeatures );
            cudaErrorCheck( hipGetLastError() );
        }
        UpdateWeight<<< uwGridDim, uwBlockDim >>>(
            dWeightArr,
            dUWSumArr,
            alpha,
            numInstances,
            numFeatures );
        cudaErrorCheck( hipGetLastError() );
    }

    cudaErrorCheck( hipDeviceSynchronize() );
    
    // hipMemcpy(weight);
    // cublasErrorCheck( hipblasDestroy( cublasHandle ) );

    time( &end );
    dif = difftime( end, start );
    printf( "Time taken is %.2lf seconds.\n", dif );

    if (uwPartSumLen > 1) hipFree( dUWPartSumArr );
    if (prePartSumLen > 1) hipFree( dPrePartSumArr );
    hipFree( dFeatureMat );
    hipFree( dFeatureMatTrans );
    hipFree( dClassArr );
    hipFree( dWeightArr );
    hipFree( dCostArr );
    hipFree( dUWSumArr );
    free( node.weights );

    return 0;
}
