#include "hip/hip_runtime.h"
#include "Helper.h"
#include "ArffImporter.h"

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>


#define WARP_SIZE 32

Node initNode( unsigned int numFeatures )
{
    Node node;
    node.numFeatures = numFeatures;
    node.weights = (float*) malloc( (numFeatures + 1) * sizeof( float ) );
    memset( node.weights, 0, (numFeatures + 1) * sizeof( float ) );

    return node;
}

void normalize(
    std::vector<NumericAttr> featureVec,
    float* featureMat,
    float* featureMatTrans,
    unsigned int numInstances )
{
    unsigned int numFeatures = featureVec.size();

    for (unsigned int i = 0; i < numFeatures; i++)
    {
        // Use either range / standard deviation
        float range = featureVec[i].max - featureVec[i].min;
        if (range == 0.0) continue;

        for (unsigned int j = 0; j < numInstances; j++)
        {
            unsigned int featureIndex = j * numFeatures + i;
            featureMat[featureIndex] =
                (featureMat[featureIndex] - featureVec[i].mean) / range;
            featureMatTrans[i * numInstances + j] = featureMat[featureIndex];
        }
    }
}

// Parallel sum combining shuffle and shared memory
__device__ __forceinline__ float parallelSum512(
    float* __restrict__ sharedData )
{
    float sum = sharedData[threadIdx.x];

    if (threadIdx.x < 256)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 256];
    __syncthreads();

    if (threadIdx.x < 128)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 128];
    __syncthreads();

    if (threadIdx.x < 64)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 64];
    __syncthreads();

#if (__CUDA_ARCH__ >= 300)
    if (threadIdx.x < 32)
    {
        sum += sharedData[threadIdx.x + 32];
        // Reduce final warp using shuffle
        // Compile unroll for loop?
        for (unsigned short shift = WARP_SIZE / 2; shift > 0; shift >>= 1)
            sum += __shfl_down( sum, shift );
    }
#else
    // fully unroll reduction within a single warp
    if (threadIdx.x < 32)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 32];
    __syncthreads();

    if (threadIdx.x < 16)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 16];
    __syncthreads();

    if (threadIdx.x < 8)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 8];
    __syncthreads();

    if (threadIdx.x < 4)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 4];
    __syncthreads();

    if (threadIdx.x < 2)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 2];
    __syncthreads();

    if (threadIdx.x < 1)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 1];
    __syncthreads();
#endif

    return sum;
}

// Parallel sum combining shuffle and shared memory
__device__ __forceinline__ float parallelSum256(
    float* __restrict__ sharedData )
{
    float sum = sharedData[threadIdx.x];

    if (threadIdx.x < 128)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 128];
    __syncthreads();

    if (threadIdx.x < 64)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 64];
    __syncthreads();

#if (__CUDA_ARCH__ >= 300)
    if (threadIdx.x < 32)
    {
        sum += sharedData[threadIdx.x + 32];
        // Reduce final warp using shuffle
        // Compile unroll for loop?
        for (unsigned short shift = WARP_SIZE / 2; shift > 0; shift >>= 1)
            sum += __shfl_down( sum, shift );
    }
#else
    // fully unroll reduction within a single warp
    if (threadIdx.x < 32)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 32];
    __syncthreads();

    if (threadIdx.x < 16)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 16];
    __syncthreads();

    if (threadIdx.x < 8)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 8];
    __syncthreads();

    if (threadIdx.x < 4)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 4];
    __syncthreads();

    if (threadIdx.x < 2)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 2];
    __syncthreads();

    if (threadIdx.x < 1)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 1];
    __syncthreads();
#endif

    return sum;
}

// Parallel sum combining shuffle and shared memory
__device__ __forceinline__ float parallelSum128(
    float* __restrict__ sharedData )
{
    float sum = sharedData[threadIdx.x];

    if (threadIdx.x < 64)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 64];
    __syncthreads();

#if (__CUDA_ARCH__ >= 300)
    if (threadIdx.x < 32)
    {
        sum += sharedData[threadIdx.x + 32];
        // Reduce final warp using shuffle
        // Compile unroll for loop?
        for (unsigned short shift = WARP_SIZE / 2; shift > 0; shift >>= 1)
            sum += __shfl_down( sum, shift );
    }
#else
    // fully unroll reduction within a single warp
    if (threadIdx.x < 32)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 32];
    __syncthreads();

    if (threadIdx.x < 16)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 16];
    __syncthreads();

    if (threadIdx.x < 8)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 8];
    __syncthreads();

    if (threadIdx.x < 4)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 4];
    __syncthreads();

    if (threadIdx.x < 2)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 2];
    __syncthreads();

    if (threadIdx.x < 1)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 1];
    __syncthreads();
#endif

    return sum;
}

// Parallel sum combining shuffle and shared memory
__device__ __forceinline__ float parallelSum64(
    float* __restrict__ sharedData )
{
    float sum = sharedData[threadIdx.x];

#if (__CUDA_ARCH__ >= 300)
    if (threadIdx.x < 32)
    {
        sum += sharedData[threadIdx.x + 32];
        // Reduce final warp using shuffle
        // Compile unroll for loop?
        for (unsigned short shift = WARP_SIZE / 2; shift > 0; shift >>= 1)
            sum += __shfl_down( sum, shift );
    }
#else
    // fully unroll reduction within a single warp
    if (threadIdx.x < 32)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 32];
    __syncthreads();

    if (threadIdx.x < 16)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 16];
    __syncthreads();

    if (threadIdx.x < 8)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 8];
    __syncthreads();

    if (threadIdx.x < 4)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 4];
    __syncthreads();

    if (threadIdx.x < 2)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 2];
    __syncthreads();

    if (threadIdx.x < 1)
        sharedData[threadIdx.x] = sum = sum + sharedData[threadIdx.x + 1];
    __syncthreads();
#endif

    return sum;
}

__device__ __forceinline__ float parallelSum32( float sum )
{
    // Reduce final warp using shuffle
    // Compile unroll for loop?
    for (unsigned short shift = WARP_SIZE / 2; shift > 0; shift >>= 1)
        sum += __shfl_down( sum, shift );

    return sum;
}

__global__ void Dot(
    float* __restrict__ dCostArr,
    const float* __restrict__ dWeightArr,
    const float* __restrict__ dFeatureMat,
    const unsigned short* __restrict__ dClassArr,
    const unsigned int numInstances,
    const unsigned int numFeatures )
{
    unsigned int instanceId = blockIdx.y * gridDim.x + blockIdx.x;
    if (instanceId >= numInstances) return;
    // if (threadIdx.x == 0) printf( "Instance ID: %u\n", instanceId );

    float dotProd = dWeightArr[numFeatures];
    const float* __restrict__ dFeaOffset = dFeatureMat + instanceId * numFeatures;

    __shared__ float sharedProd[512];
    unsigned int offset = threadIdx.x * 2;
    float partialSum = 0.0f;
    if (offset < numFeatures)
        partialSum += dWeightArr[offset] * dFeaOffset[offset];
    // else return;
    if (offset + 1 < numFeatures)
        partialSum += dWeightArr[offset + 1] * dFeaOffset[offset + 1];
    sharedProd[threadIdx.x] = partialSum;
    __syncthreads();

    dotProd += parallelSum512( sharedProd );
    if (threadIdx.x == 0) dCostArr[instanceId] = dotProd;
}

__global__ void ComputeCost(
    float* __restrict__ dCostArr,
    const unsigned short* __restrict__ dClassArr,
    const unsigned int numInstances )
{
    unsigned int instanceId = blockIdx.x * blockDim.x + threadIdx.x;
    if (instanceId >= numInstances) return;

    float cost = dCostArr[instanceId];
    cost = 1.0 / (1.0 + exp(-cost)) - (float) dClassArr[instanceId];
    dCostArr[instanceId] = cost;
}

__global__ void UpdateWeightL2(
    float* __restrict__ dWeightArr,
    const float* __restrict__ dPartSumArr,
    const unsigned int alpha,
    const unsigned int partSumLen,
    const unsigned int numInstances,
    const unsigned int numFeatures )
{
    if (blockIdx.x >= numFeatures) return;

    float sum = 0.0f;
    if (blockDim.x == 32)
    {
        if (threadIdx.x < partSumLen)
            sum = dPartSumArr[blockIdx.x * partSumLen + threadIdx.x];
        sum = parallelSum32( sum );
    }
    else
    {
        extern __shared__ float sharedPartSum[];
        unsigned int offset = threadIdx.x * 2;
        if (offset < partSumLen)
            sum += dPartSumArr[offset];
        if (offset + 1 < partSumLen)
            sum += dPartSumArr[offset + 1];
        sharedPartSum[threadIdx.x] = sum;
        __syncthreads();

        switch (blockDim.x)
        {
            case 64:
                sum = parallelSum64( sharedPartSum );
                break;
            case 128:
                sum = parallelSum128( sharedPartSum );
                break;
            case 256:
                sum = parallelSum256( sharedPartSum );
                break;
            case 512:
                sum = parallelSum512( sharedPartSum );
                break;
            default:
                break;
        }
    }

    // Update weights
    if (threadIdx.x == 0)
    {
        dWeightArr[blockIdx.x] -=
            alpha / (float) numInstances * sum;

        if (blockIdx.x == 0)
            printf( "Updating weights completed, weight: %f\n", dWeightArr[0] );
    }
}

__global__ void UpdateWeight(
    float* __restrict__ dWeightArr,
    float* __restrict__ dPartSumArr,
    const float* __restrict__ dCostArr,
    const float* __restrict__ dFeatureMatTrans,
    const unsigned int alpha,
    const unsigned int partSumLen,
    const unsigned int numInstances,
    const unsigned int numFeatures )
{
    unsigned int featureId;
    unsigned int offset;
    if (partSumLen == 1)
    {
        featureId = blockIdx.x;
        offset = threadIdx.x * 2;
    }
    else
    {
        featureId = blockIdx.y;
        offset = blockIdx.x * blockDim.x * 2 + threadIdx.x * 2;
    }

    if (featureId >= numFeatures) return;

    const float* __restrict__ dFeaMatTransOffset =
        dFeatureMatTrans + featureId * numInstances;
    float partialSum = 0.0;

    if (partSumLen == 1)
    {
        if (blockDim.x == 32)
        {
            if (threadIdx.x < partSumLen)
                partialSum =
                    dFeaMatTransOffset[offset] *
                    dCostArr[offset];
            partialSum = parallelSum32( partialSum );
        }
        else
        {
            extern __shared__ float sharedPartSum[];
            if (offset < partSumLen)
                partialSum += dFeaMatTransOffset[offset] *
                    dCostArr[offset];
            if (offset + 1 < partSumLen)
                partialSum += dFeaMatTransOffset[offset + 1] *
                    dCostArr[offset + 1];
            sharedPartSum[threadIdx.x] = partialSum;
            __syncthreads();

            switch (blockDim.x)
            {
                case 64:
                    partialSum = parallelSum64( sharedPartSum );
                    break;
                case 128:
                    partialSum = parallelSum128( sharedPartSum );
                    break;
                case 256:
                    partialSum = parallelSum256( sharedPartSum );
                    break;
                case 512:
                    partialSum = parallelSum512( sharedPartSum );
                    break;
                default:
                    break;
            }
        }
    }
    else
    {
        extern __shared__ float sharedPartSum[];
        if (offset < numInstances)
            partialSum += dFeaMatTransOffset[offset] * dCostArr[offset];
        if (offset + 1 < numInstances)
            partialSum += dFeaMatTransOffset[offset + 1] * dCostArr[offset + 1];
        sharedPartSum[threadIdx.x] = partialSum;
        __syncthreads();

        partialSum = parallelSum512( sharedPartSum );
        if (threadIdx.x == 0)
            dPartSumArr[featureId * partSumLen + blockIdx.x] = partialSum;
    }

    // const float* __restrict__ dFeaMatTransOffset =
    //     dFeatureMatTrans + featureId * numInstances;
    // __shared__ float sharedProd[512];
    // float partialSum = 0.0;
    // if (offset < numInstances)
    //     partialSum += dFeaMatTransOffset[offset] * dCostArr[offset];
    // if (offset + 1 < numInstances)
    //     partialSum += dFeaMatTransOffset[offset + 1] * dCostArr[offset + 1];
    // sharedProd[threadIdx.x] = partialSum;
    // __syncthreads();

    // partialSum = parallelSum512( sharedProd );
    // if (threadIdx.x == 0)
    //     dPartSumArr[featureId * partSumLen + blockIdx.x] = partialSum;

    // // Update weights
    if (partSumLen == 1 && threadIdx.x == 0)
    {
        dWeightArr[featureId] -=
            alpha / (float) numInstances * partialSum;

        if (featureId == 0)
            printf( "Updating weights completed, weight: %f\n", partialSum );
    }
}

inline void cudaErrorCheck( hipError_t cudaRes )
{
    if (cudaRes != hipSuccess)
        printf(
            "kernel launch failed with error \"%s\".\n",
            hipGetErrorString( cudaRes )
        );
}

int main()
{
    ArffImporter trainSetImporter;
    trainSetImporter.Read( "Dataset/train/train-first1000.arff" );

    // ArffImporter testSetImporter;
    // testSetImporter.Read( "Dataset/test/dev-first1000.arff" );

    unsigned int numInstances = trainSetImporter.GetNumInstances();
    float* featureMat = trainSetImporter.GetFeatureMat();
    float* featureMatTrans = trainSetImporter.GetFeatureMatTrans();
    unsigned short* classArr = trainSetImporter.GetClassIndex();
    std::vector<NumericAttr> featureVec = trainSetImporter.GetFeatures();
    unsigned int numFeatures = featureVec.size();

    normalize( featureVec, featureMat, featureMatTrans, numInstances );
    Node node = initNode( numFeatures );

    /*----------- Determine block and grid size of Dot kernel -----------*/
    dim3 dotBlockDim;
    dim3 dotGridDim;
    // Assume numFeatures <= 1024 (max number of threads per block)
    // dotBlockDim.x = numFeatures;
    dotBlockDim.x = 512;
    if (numInstances < 1024)
        dotGridDim.x = numInstances;
    else
    {
        dotGridDim.x = 1024;
        dotGridDim.y = (numInstances + dotGridDim.x - 1) / dotGridDim.x;
    }

    /*------- Determine block and grid size of ComputeCost kernel -------*/
    dim3 ccBlockDim;
    dim3 ccGridDim;
    if (numInstances > 1024)
    {
        ccBlockDim.x = 1024;
        ccGridDim.x = (numInstances + 1023) / 1024;
    }
    else ccBlockDim.x = numInstances;

    /*------- Determine block and grid size of UpdateWeight kernel -------*/
    dim3 uwBlockDimL1;
    dim3 uwGridDimL1;
    dim3 uwBlockDimL2;
    dim3 uwGridDimL2;
    unsigned int partSumLen;
    unsigned int sharedMemoSizeL1;
    unsigned int sharedMemoSizeL2;
    // Assume numFeatures < 1024
    if (numInstances > 1024)
    {
        uwBlockDimL1.x = 512;
        uwGridDimL1.x = (numInstances + 1023) / 1024;
        uwGridDimL1.y = numFeatures;
        partSumLen = uwGridDimL1.x;
        sharedMemoSizeL1 = 512 * sizeof( float );

        uwGridDimL2.x = numFeatures;
        // Assume partSumLen <= 1024
        if (partSumLen <= 32)
        {
            uwBlockDimL2.x = 32;
            sharedMemoSizeL2 = 0;
        }
        else
        {
            if (partSumLen <= 64) uwBlockDimL2.x = 32;
            else if (partSumLen <= 128) uwBlockDimL2.x = 64;
            else if (partSumLen <= 256) uwBlockDimL2.x = 128;
            else if (partSumLen <= 512) uwBlockDimL2.x = 256;
            else uwBlockDimL2.x = 512;
            sharedMemoSizeL2 = uwBlockDimL2.x * sizeof( float );
        }
    }
    else
    {
        partSumLen = 1;
        uwGridDimL1.x = numFeatures;
        if (numInstances <= 32)
        {
            uwBlockDimL1.x = 32;
            sharedMemoSizeL1 = 0;
        }
        else
        {
            if (numInstances <= 64) uwBlockDimL1.x = 32;
            else if (numInstances <= 128) uwBlockDimL1.x = 64;
            else if (numInstances <= 256) uwBlockDimL1.x = 128;
            else if (numInstances <= 512) uwBlockDimL1.x = 256;
            sharedMemoSizeL1 = uwBlockDimL1.x * sizeof( float );
        }
    }

    float* dCostArr;
    float* dWeightArr;
    float* dFeatureMat;
    float* dFeatureMatTrans;
    float* dPartSumArr;
    unsigned short* dClassArr;
    cudaErrorCheck( hipMalloc( (void**) &dWeightArr, (numFeatures + 1) * sizeof( float ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dCostArr, numInstances * sizeof( float ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dFeatureMat, numInstances * numFeatures * sizeof( float ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dFeatureMatTrans, numInstances * numFeatures * sizeof( float ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dClassArr, numInstances * sizeof( unsigned short ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dPartSumArr, partSumLen * numFeatures * sizeof( float ) ) );
    cudaErrorCheck( hipMemcpyAsync(
        dFeatureMat,
        featureMat,
        numInstances * numFeatures * sizeof( float ),
        hipMemcpyHostToDevice ) );
    cudaErrorCheck( hipMemcpyAsync(
        dFeatureMatTrans,
        featureMatTrans,
        numInstances * numFeatures * sizeof( float ),
        hipMemcpyHostToDevice ) );
    cudaErrorCheck( hipMemcpyAsync(
        dWeightArr,
        node.weights,
        (numFeatures + 1) * sizeof( float ),
        hipMemcpyHostToDevice ) );
    cudaErrorCheck( hipMemcpyAsync(
        dClassArr,
        classArr,
        numInstances * sizeof( unsigned short ),
        hipMemcpyHostToDevice ) );

    // Gradient descent params
    unsigned int alpha = 50;
    unsigned int maxIter = 200;
    unsigned int iter = 0;

    time_t start, end;
    float dif;
    time( &start );
    
    printf( "\nStart gradient descent...\n" );

    // Gradient descent
    while (iter++ < maxIter)
    {
        Dot<<< dotGridDim, dotBlockDim >>>(
            dCostArr,
            dWeightArr,
            dFeatureMat,
            dClassArr,
            numInstances,
            numFeatures );
        cudaErrorCheck( hipGetLastError() );

        ComputeCost<<< ccGridDim, ccBlockDim >>>(
            dCostArr,
            dClassArr,
            numInstances );
        cudaErrorCheck( hipGetLastError() );

        UpdateWeight<<< uwGridDimL1, uwBlockDimL1, sharedMemoSizeL1 >>>(
            dWeightArr,
            dPartSumArr,
            dCostArr,
            dFeatureMatTrans,
            alpha,
            partSumLen,
            numInstances,
            numFeatures );
        cudaErrorCheck( hipGetLastError() );
        if (partSumLen > 1)
        {
            UpdateWeightL2<<< uwGridDimL2, uwBlockDimL2, sharedMemoSizeL2 >>>(
                dWeightArr,
                dPartSumArr,
                alpha,
                partSumLen,
                numInstances,
                numFeatures );
            cudaErrorCheck( hipGetLastError() );
        }
    }

    cudaErrorCheck( hipDeviceSynchronize() );
    
    // hipMemcpy(weight);
    // cublasErrorCheck( hipblasDestroy( cublasHandle ) );

    time( &end );
    dif = difftime( end, start );
    printf( "Time taken is %.2lf seconds.\n", dif );

    hipFree( dFeatureMat );
    hipFree( dFeatureMatTrans );
    hipFree( dClassArr );
    hipFree( dWeightArr );
    hipFree( dCostArr );
    hipFree( dPartSumArr );
    free( node.weights );

    return 0;
}
