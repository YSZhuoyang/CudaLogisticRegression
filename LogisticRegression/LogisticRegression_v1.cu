#include "hip/hip_runtime.h"
#include "Helper.h"
#include "ArffImporter.h"

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
// #include "hipblas.h"


Node initNode( unsigned int numFeatures )
{
    Node node;
    node.numFeatures = numFeatures;
    node.weights = (double*) malloc( (numFeatures + 1) * sizeof( double ) );
    memset( node.weights, 0, (numFeatures + 1) * sizeof( double ) );

    return node;
}

void normalize(
    std::vector<NumericAttr> featureVec,
    double* featureBuff,
    unsigned int numInstances )
{
    unsigned int numFeatures = featureVec.size();

    for (unsigned int i = 0; i < numFeatures; i++)
    {
        // Use either range / standard deviation
        double range = featureVec[i].max - featureVec[i].min;
        if (range == 0.0) continue;

        for (unsigned int j = 0; j < numInstances; j++)
        {
            unsigned int featureIndex = j * numFeatures + i;
            featureBuff[featureIndex] =
                (featureBuff[featureIndex] - featureVec[i].mean) / range;
        }
    }
}

__device__ double activate(
    Node* node,
    double* inputArr )
{
    double linearRes = node->weights[node->numFeatures];
    node->inputs = inputArr;

    unsigned int numFeatures = node->numFeatures;
    for (unsigned int i = 0; i < numFeatures; i++)
        linearRes += node->weights[i] * node->inputs[i];

    node->output = 1.0 / (1.0 + exp(-linearRes));

    return node->output;
}

__device__ __forceinline__ void parallelSum(
    double* dProductArr,
    const unsigned int elementId,
    const unsigned int length )
{
    for (unsigned int i = length; i > 1; i /= 2)
    {
        if (elementId < i / 2)
        {
            dProductArr[elementId] +=
                dProductArr[elementId + i / 2];

            // Odd
            if (i & 1 && elementId == i / 2 - 1)
                dProductArr[elementId] += dProductArr[i - 1];
        }
        __syncthreads();
    }
}

__global__ void Activate(
    double* dDiffArr,
    double* dWeightArr,
    const double* dFeatureBuff,
    const unsigned short* dClassBuff,
    const unsigned int numInstances,
    const unsigned int numFeatures )
{
    unsigned int instanceId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int featureId = threadIdx.y * blockDim.x + threadIdx.x;
    if (instanceId >= numInstances || featureId >= numFeatures) return;
    // if (featureId == 0) printf( "Instance ID: %u\n", instanceId );

    double hRes = dWeightArr[numFeatures];
    extern __shared__ double dProductShared[];
    dProductShared[featureId] =
        dWeightArr[featureId] * dFeatureBuff[instanceId * numFeatures + featureId];
    __syncthreads();

    // Assume numFeatures is big
    parallelSum( dProductShared, featureId, numFeatures );

    if (featureId == 0)
    {
        hRes += dProductShared[0];
        hRes = 1.0 / (1.0 + exp(-hRes));
        dDiffArr[instanceId] = hRes - (double) dClassBuff[instanceId];
    }
}

__global__ void UpdateWeight(
    double* dDiffArr,
    double* dWeightArr,
    const double* dFeatureBuff,
    const unsigned int alpha,
    const unsigned int chunkSize,
    const unsigned int numInstances,
    const unsigned int numFeatures )
{
    // One block per feature, one thread per group of instances
    unsigned int featureId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int instChunkId = threadIdx.y * blockDim.x + threadIdx.x;
    if (instChunkId >= numInstances || featureId >= numFeatures) return;

    unsigned int stopId;
    // Last chunk
    if (instChunkId == blockDim.x - 1)
        stopId = numInstances;
    else
        stopId = chunkSize * (instChunkId + 1);

    double multSum = 0.0;
    // Values of one feature
    extern __shared__ double dProductShared[];
    dProductShared[instChunkId] = 0.0;
    for (unsigned int i = chunkSize * instChunkId; i < stopId; i++)
        multSum += dFeatureBuff[i * numFeatures + featureId] * dDiffArr[i];
    dProductShared[instChunkId] = multSum;
    __syncthreads();

    // Parallel sum
    // Assume numInstances is big
    parallelSum( dProductShared, instChunkId, blockDim.x );

    // Update weights
    if (instChunkId == 0)
    {
        dWeightArr[featureId] -=
            alpha / (double) numInstances * dProductShared[0];

        if (featureId == 0)
            printf( "Updating weights completed, weight: %f\n", dWeightArr[0] );
    }
}

inline void cudaErrorCheck( hipError_t cudaRes )
{
    if (cudaRes != hipSuccess)
        printf(
            "kernel launch failed with error \"%s\".\n",
            hipGetErrorString( cudaRes )
        );
}

// void cublasErrorCheck( hipblasStatus_t cublasRes )
// {
//     if (cublasRes != HIPBLAS_STATUS_SUCCESS)
//         printf( "Cublas library failed to load.\n" );
// }

int main()
{
    ArffImporter trainSetImporter;
    trainSetImporter.Read( "Dataset/train/train-first1000.arff" );

    // ArffImporter testSetImporter;
    // testSetImporter.Read( "Dataset/test/dev-first1000.arff" );

    unsigned int numInstances = trainSetImporter.GetNumInstances();
    double* featureBuff = trainSetImporter.GetInstances();
    unsigned short* classIndexBuff = trainSetImporter.GetClassIndex();
    std::vector<NumericAttr> featureVec = trainSetImporter.GetFeatures();
    unsigned int numFeatures = featureVec.size();

    unsigned int alpha = 50;
    unsigned int maxIter = 200;
    unsigned int iter = 0;

    // Determine block and grid size
    dim3 actBlockDim;
    dim3 actGridDim;
    dim3 sumCostBlockDim;
    dim3 uwBlockDim;
    dim3 uwGridDim;
    if (numInstances < 1024)
    {
        actGridDim.x = numInstances;
        sumCostBlockDim.x = numInstances;
    }
    else
    {
        actGridDim.x = 1024;
        actGridDim.y = (numInstances + actGridDim.x - 1) / actGridDim.x;
        sumCostBlockDim.x = 1024;
        sumCostBlockDim.y = (numInstances + sumCostBlockDim.x - 1) / sumCostBlockDim.x;
    }

    if (numFeatures < 1024) actBlockDim.x = numFeatures;
    else
    {
        actBlockDim.x = 1024;
        actBlockDim.y = (numFeatures + actBlockDim.x - 1) / actBlockDim.x;
    }

    uwBlockDim.x = 1000;
    uwGridDim = actBlockDim;
    unsigned int uwChunkSize = numInstances / uwBlockDim.x;

    normalize( featureVec, featureBuff, numInstances );
    Node node = initNode( numFeatures );

    double* dDiffArr;
    double* dWeightArr;
    double* dFeatureBuff;
    unsigned short* dClassBuff;
    cudaErrorCheck( hipMalloc( (void**) &dWeightArr, (numFeatures + 1) * sizeof( double ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dDiffArr, numInstances * sizeof( double ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dFeatureBuff, numInstances * numFeatures * sizeof( double ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dClassBuff, numInstances * sizeof( unsigned short ) ) );

    cudaErrorCheck( hipMemcpy(
        dFeatureBuff,
        featureBuff,
        numInstances * numFeatures * sizeof( double ),
        hipMemcpyHostToDevice ) );
    cudaErrorCheck( hipMemcpy(
        dWeightArr,
        node.weights,
        (numFeatures + 1) * sizeof( double ),
        hipMemcpyHostToDevice ) );
    cudaErrorCheck( hipMemcpy(
        dClassBuff,
        classIndexBuff,
        numInstances * sizeof( unsigned short ),
        hipMemcpyHostToDevice ) );

    // hipblasHandle_t cublasHandle;
    // cublasErrorCheck( hipblasCreate( &cublasHandle ) );

    time_t start, end;
    double dif;
    time( &start );
    
    printf( "\nStart gradient descent...\n" );

    // Gradient descent
    do
    {
        Activate<<< actGridDim, actBlockDim, numFeatures * sizeof( double ) >>>(
            dDiffArr,
            dWeightArr,
            dFeatureBuff,
            dClassBuff,
            numInstances,
            numFeatures );
        cudaErrorCheck( hipGetLastError() );

        UpdateWeight<<< uwGridDim, uwBlockDim, uwBlockDim.x * sizeof( double ) >>>(
            dDiffArr,
            dWeightArr,
            dFeatureBuff,
            alpha,
            uwChunkSize,
            numInstances,
            numFeatures );
        cudaErrorCheck( hipGetLastError() );

        iter++;
    }
    // while (iter == 1 || (deltaCostSum > 1.0 && iter < maxIter));
    while (iter == 1 || iter < maxIter);

    cudaErrorCheck( hipDeviceSynchronize() );
    
    // hipMemcpy(weight);
    // cublasErrorCheck( hipblasDestroy( cublasHandle ) );

    time( &end );
    dif = difftime( end, start );

    printf( "Time taken is %.2lf seconds.\n", dif );

    hipFree( dFeatureBuff );
    hipFree( dClassBuff );
    hipFree( dWeightArr );
    hipFree( dDiffArr );

    free( node.weights );

    return 0;
}
