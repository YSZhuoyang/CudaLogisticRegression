#include "hip/hip_runtime.h"
#include "Helper.h"
#include "ArffImporter.h"

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
// #include "hipblas.h"


Node initNode( unsigned int numFeatures )
{
    Node node;
    node.numFeatures = numFeatures;
    node.weights = (double*) malloc( (numFeatures + 1) * sizeof( double ) );
    memset( node.weights, 0, (numFeatures + 1) * sizeof( double ) );

    return node;
}

void normalize(
    std::vector<NumericAttr> featureVec,
    double* featureBuff,
    double* featureBuffTrans,
    unsigned int numInstances )
{
    unsigned int numFeatures = featureVec.size();

    for (unsigned int i = 0; i < numFeatures; i++)
    {
        // Use either range / standard deviation
        double range = featureVec[i].max - featureVec[i].min;
        if (range == 0.0) continue;

        for (unsigned int j = 0; j < numInstances; j++)
        {
            unsigned int featureIndex = j * numFeatures + i;
            featureBuff[featureIndex] =
                (featureBuff[featureIndex] - featureVec[i].mean) / range;
            featureBuffTrans[i * numInstances + j] = featureBuff[featureIndex];
        }
    }
}

__device__ __forceinline__ void parallelSum(
    double* sharedData,
    const unsigned int elementId,
    const unsigned int length )
{
    for (unsigned int i = length; i > 1; i >>= 1)
    {
        unsigned int shift = i / 2;
        if (elementId < shift)
        {
            sharedData[elementId] +=
                sharedData[elementId + shift];

            // Odd
            if (i & 1 && elementId == shift - 1)
                sharedData[elementId] += sharedData[i - 1];
        }
        __syncthreads();
    }
}

__global__ void Activate(
    double* dDiffArr,
    double* dWeightArr,
    const double* dFeatureBuff,
    const unsigned short* dClassBuff,
    const unsigned int numInstances,
    const unsigned int numFeatures )
{
    unsigned int instanceId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int featureId = threadIdx.y * blockDim.x + threadIdx.x;
    if (instanceId >= numInstances || featureId >= numFeatures) return;
    // if (featureId == 0) printf( "Instance ID: %u\n", instanceId );

    double hRes = dWeightArr[numFeatures];
    const double* dFeaOffset = dFeatureBuff + instanceId * numFeatures;
    extern __shared__ double dProductShared[];
    dProductShared[featureId] =
        dWeightArr[featureId] * dFeaOffset[featureId];
    __syncthreads();

    // Assume numFeatures is big
    parallelSum( dProductShared, featureId, numFeatures );

    if (featureId == 0)
    {
        hRes += dProductShared[0];
        hRes = 1.0 / (1.0 + exp(-hRes));
        dDiffArr[instanceId] = hRes - (double) dClassBuff[instanceId];
    }
}

__global__ void UpdateWeight(
    double* dDiffArr,
    double* dWeightArr,
    const double* dFeatureBuffTrans,
    const unsigned int alpha,
    const unsigned int chunkSize,
    const unsigned int numInstances,
    const unsigned int numFeatures )
{
    // One block per feature, one thread per group of instances
    unsigned int featureId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int instChunkId = threadIdx.y * blockDim.x + threadIdx.x;
    if (instChunkId >= numInstances || featureId >= numFeatures) return;

    unsigned int stopId;
    // Last chunk
    if (instChunkId == blockDim.x - 1)
        stopId = numInstances;
    else
        stopId = chunkSize * (instChunkId + 1);

    double multSum = 0.0;
    // Values of one feature
    extern __shared__ double dProductShared[];
    for (unsigned int i = chunkSize * instChunkId; i < stopId; i++)
        multSum += dFeatureBuffTrans[featureId * numInstances + i] * dDiffArr[i];
    dProductShared[instChunkId] = multSum;
    __syncthreads();

    // Assume numInstances is big
    parallelSum( dProductShared, instChunkId, blockDim.x );

    // Update weights
    if (instChunkId == 0)
    {
        dWeightArr[featureId] -=
            alpha / (double) numInstances * dProductShared[0];

        if (featureId == 0)
            printf( "Updating weights completed, weight: %f\n", dWeightArr[0] );
    }
}

inline void cudaErrorCheck( hipError_t cudaRes )
{
    if (cudaRes != hipSuccess)
        printf(
            "kernel launch failed with error \"%s\".\n",
            hipGetErrorString( cudaRes )
        );
}

// void cublasErrorCheck( hipblasStatus_t cublasRes )
// {
//     if (cublasRes != HIPBLAS_STATUS_SUCCESS)
//         printf( "Cublas library failed to load.\n" );
// }

int main()
{
    ArffImporter trainSetImporter;
    trainSetImporter.Read( "Dataset/train/train-first1000.arff" );

    // ArffImporter testSetImporter;
    // testSetImporter.Read( "Dataset/test/dev-first1000.arff" );

    unsigned int numInstances = trainSetImporter.GetNumInstances();
    double* featureBuff = trainSetImporter.GetFeatureBuff();
    double* featureBuffTrans = trainSetImporter.GetFeatureBuffTrans();
    unsigned short* classIndexBuff = trainSetImporter.GetClassIndex();
    std::vector<NumericAttr> featureVec = trainSetImporter.GetFeatures();
    unsigned int numFeatures = featureVec.size();

    unsigned int alpha = 50;
    unsigned int maxIter = 200;
    unsigned int iter = 0;

    // Determine block and grid size of Activat kernel
    dim3 actBlockDim;
    dim3 actGridDim;
    dim3 uwBlockDim;
    dim3 uwGridDim;
    // Assume numFeatures <= 1024 (max number of threads per block)
    actBlockDim.x = numFeatures;
    if (numInstances < 1024)
        actGridDim.x = numInstances;
    else
    {
        actGridDim.x = 1024;
        actGridDim.y = (numInstances + actGridDim.x - 1) / actGridDim.x;
    }

    // Determine block and grid size of UpdateWeight kernel
    uwBlockDim.x = actGridDim.x;
    uwGridDim = actBlockDim;
    unsigned int uwChunkSize = numInstances / uwBlockDim.x;

    normalize( featureVec, featureBuff, featureBuffTrans, numInstances );
    Node node = initNode( numFeatures );

    double* dDiffArr;
    double* dWeightArr;
    double* dFeatureBuff;
    double* dFeatureBuffTrans;
    unsigned short* dClassBuff;
    cudaErrorCheck( hipMalloc( (void**) &dWeightArr, (numFeatures + 1) * sizeof( double ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dDiffArr, numInstances * sizeof( double ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dFeatureBuff, numInstances * numFeatures * sizeof( double ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dFeatureBuffTrans, numInstances * numFeatures * sizeof( double ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dClassBuff, numInstances * sizeof( unsigned short ) ) );

    cudaErrorCheck( hipMemcpy(
        dFeatureBuff,
        featureBuff,
        numInstances * numFeatures * sizeof( double ),
        hipMemcpyHostToDevice ) );
    cudaErrorCheck( hipMemcpy(
        dFeatureBuffTrans,
        featureBuffTrans,
        numInstances * numFeatures * sizeof( double ),
        hipMemcpyHostToDevice ) );
    cudaErrorCheck( hipMemcpy(
        dWeightArr,
        node.weights,
        (numFeatures + 1) * sizeof( double ),
        hipMemcpyHostToDevice ) );
    cudaErrorCheck( hipMemcpy(
        dClassBuff,
        classIndexBuff,
        numInstances * sizeof( unsigned short ),
        hipMemcpyHostToDevice ) );

    // hipblasHandle_t cublasHandle;
    // cublasErrorCheck( hipblasCreate( &cublasHandle ) );

    time_t start, end;
    double dif;
    time( &start );
    
    printf( "\nStart gradient descent...\n" );

    // Gradient descent
    do
    {
        Activate<<< actGridDim, actBlockDim, numFeatures * sizeof( double ) >>>(
            dDiffArr,
            dWeightArr,
            dFeatureBuff,
            dClassBuff,
            numInstances,
            numFeatures );
        cudaErrorCheck( hipGetLastError() );

        UpdateWeight<<< uwGridDim, uwBlockDim, uwBlockDim.x * sizeof( double ) >>>(
            dDiffArr,
            dWeightArr,
            dFeatureBuffTrans,
            alpha,
            uwChunkSize,
            numInstances,
            numFeatures );
        cudaErrorCheck( hipGetLastError() );

        iter++;
    }
    while (iter == 1 || iter < maxIter);

    cudaErrorCheck( hipDeviceSynchronize() );
    
    // hipMemcpy(weight);
    // cublasErrorCheck( hipblasDestroy( cublasHandle ) );

    time( &end );
    dif = difftime( end, start );
    printf( "Time taken is %.2lf seconds.\n", dif );

    hipFree( dFeatureBuff );
    hipFree( dFeatureBuffTrans );
    hipFree( dClassBuff );
    hipFree( dWeightArr );
    hipFree( dDiffArr );

    free( node.weights );

    return 0;
}
