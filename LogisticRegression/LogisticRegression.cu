#include "hip/hip_runtime.h"
#include "Helper.h"
#include "ArffImporter.h"

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>


#define WARP_SIZE 32

Node initNode( unsigned int numFeatures )
{
    Node node;
    node.numFeatures = numFeatures;
    node.weights = (float*) malloc( (numFeatures + 1) * sizeof( float ) );
    memset( node.weights, 0, (numFeatures + 1) * sizeof( float ) );

    return node;
}

void normalize(
    std::vector<NumericAttr> featureVec,
    float* featureMat,
    float* featureMatTrans,
    unsigned int numInstances )
{
    unsigned int numFeatures = featureVec.size();

    for (unsigned int i = 0; i < numFeatures; i++)
    {
        // Use either range / standard deviation
        float range = featureVec[i].max - featureVec[i].min;
        if (range == 0.0) continue;

        for (unsigned int j = 0; j < numInstances; j++)
        {
            unsigned int featureIndex = j * numFeatures + i;
            featureMat[featureIndex] =
                (featureMat[featureIndex] - featureVec[i].mean) / range;
            featureMatTrans[i * numInstances + j] = featureMat[featureIndex];
        }
    }
}

__device__ __forceinline__ float shuffleReduceSum( float regValue )
{
    for (unsigned int shift = WARP_SIZE / 2; shift > 0; shift >>= 1)
        regValue += __shfl_down( regValue, shift );
    // for (unsigned int i = 1; i < WARP_SIZE; i *= 2) // i =<< 1
    //     regValue += __shfl_xor( regValue, i );
    return regValue;
}

// Sum up any arrays with a maximum length of 1024
__device__ __forceinline__ float shuffleParallelSum(
    float regValue,
    const unsigned int numWarps )
{
    __shared__ float shared[32];
    int warpThreadId = threadIdx.x % WARP_SIZE;
    int warpId = threadIdx.x / WARP_SIZE;

    // Performing warp reduction. Only the threads with 0 index
    // within the warp have the "val" value set with the warp reduction result
    regValue = shuffleReduceSum( regValue );

    // Only the threads with 0 index within the warp write the warp result to shared memory
    if (warpThreadId == 0) shared[warpId] = regValue;

    // Wait for all warp reductions
    __syncthreads();

    // There will be at most 1024 threads within a block.
    // The partial sum is read from shared memory only the corresponding
    // warp existed, otherwise the partial sum is set to zero.
    if (threadIdx.x < numWarps)
    {
        regValue = shared[warpThreadId];
        // The first warp performs the final partial warp summation.
        // Note that numWarps is always smaller than 32 given an array with a maximum length of 1024.
        if (warpId == 0) return shuffleReduceSum( regValue );
    }

    return 0;
}

// Parallel sum using a shared memory
__device__ __forceinline__ void parallelSum(
    float* __restrict__ sharedData,
    const unsigned int length )
{
    for (unsigned int i = length; i > 1; i >>= 1)
    {
        unsigned int shift = i / 2;
        if (threadIdx.x < shift)
        {
            sharedData[threadIdx.x] +=
                sharedData[threadIdx.x + shift];

            // Odd
            if (i & 1 && threadIdx.x == shift - 1)
                sharedData[threadIdx.x] += sharedData[i - 1];
        }
        __syncthreads();
    }
}

__global__ void Dot(
    float* __restrict__ dCostArr,
    const float* __restrict__ dWeightArr,
    const float* __restrict__ dFeatureMat,
    const unsigned short* __restrict__ dClassArr,
    const unsigned int numWarps,
    const unsigned int numInstances,
    const unsigned int numFeatures )
{
    unsigned int instanceId = blockIdx.y * gridDim.x + blockIdx.x;
    // unsigned int featureId = threadIdx.y * blockDim.x + threadIdx.x;
    if (instanceId >= numInstances || threadIdx.x >= numFeatures) return;
    // if (threadIdx.x == 0) printf( "Instance ID: %u\n", instanceId );

    float dotProd = dWeightArr[numFeatures];
    const float* __restrict__ dFeaOffset = dFeatureMat + instanceId * numFeatures;

    dotProd += shuffleParallelSum(
        dWeightArr[threadIdx.x] * dFeaOffset[threadIdx.x],
        numWarps );

    if (threadIdx.x == 0) dCostArr[instanceId] = dotProd;
}

__global__ void ComputeCost(
    float* __restrict__ dCostArr,
    const unsigned short* __restrict__ dClassArr,
    const unsigned int numInstances )
{
    unsigned int instanceId = blockIdx.x * blockDim.x + threadIdx.x;
    if (instanceId >= numInstances) return;

    float cost = dCostArr[instanceId];
    cost = 1.0 / (1.0 + exp(-cost)) - (float) dClassArr[instanceId];
    dCostArr[instanceId] = cost;
}

__global__ void UpdateWeight(
    float* __restrict__ dWeightArr,
    const float* __restrict__ dCostArr,
    const float* __restrict__ dFeatureMatTrans,
    const unsigned int alpha,
    const unsigned int chunkSize,
    const unsigned int numWarps,
    const unsigned int numInstances,
    const unsigned int numFeatures )
{
    // One block per feature, one thread per group of instances
    unsigned int featureId = blockIdx.y * gridDim.x + blockIdx.x;
    // unsigned int instChunkId = threadIdx.y * blockDim.x + threadIdx.x;
    if (threadIdx.x >= numInstances || featureId >= numFeatures) return;

    unsigned int stopId;
    if (threadIdx.x == blockDim.x - 1) // Last chunk
        stopId = numInstances;
    else
        stopId = chunkSize * (threadIdx.x + 1);

    float multSum = 0.0;
    for (unsigned int i = chunkSize * threadIdx.x; i < stopId; i++)
        multSum += dFeatureMatTrans[featureId * numInstances + i] * dCostArr[i];
    multSum = shuffleParallelSum(
        multSum,
        numWarps );

    // Update weights
    if (threadIdx.x == 0)
    {
        dWeightArr[featureId] -=
            alpha / (float) numInstances * multSum;

        if (featureId == 0)
            printf( "Updating weights completed, weight: %f\n", dWeightArr[0] );
    }
}

inline void cudaErrorCheck( hipError_t cudaRes )
{
    if (cudaRes != hipSuccess)
        printf(
            "kernel launch failed with error \"%s\".\n",
            hipGetErrorString( cudaRes )
        );
}

int main()
{
    ArffImporter trainSetImporter;
    trainSetImporter.Read( "Dataset/train/train-first1000.arff" );

    // ArffImporter testSetImporter;
    // testSetImporter.Read( "Dataset/test/dev-first1000.arff" );

    unsigned int numInstances = trainSetImporter.GetNumInstances();
    float* featureMat = trainSetImporter.GetFeatureMat();
    float* featureMatTrans = trainSetImporter.GetFeatureMatTrans();
    unsigned short* classArr = trainSetImporter.GetClassIndex();
    std::vector<NumericAttr> featureVec = trainSetImporter.GetFeatures();
    unsigned int numFeatures = featureVec.size();

    normalize( featureVec, featureMat, featureMatTrans, numInstances );
    Node node = initNode( numFeatures );

    float* dCostArr;
    float* dWeightArr;
    float* dFeatureMat;
    float* dFeatureMatTrans;
    unsigned short* dClassArr;
    cudaErrorCheck( hipMalloc( (void**) &dWeightArr, (numFeatures + 1) * sizeof( float ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dCostArr, numInstances * sizeof( float ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dFeatureMat, numInstances * numFeatures * sizeof( float ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dFeatureMatTrans, numInstances * numFeatures * sizeof( float ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dClassArr, numInstances * sizeof( unsigned short ) ) );
    cudaErrorCheck( hipMemcpyAsync(
        dFeatureMat,
        featureMat,
        numInstances * numFeatures * sizeof( float ),
        hipMemcpyHostToDevice ) );
    cudaErrorCheck( hipMemcpyAsync(
        dFeatureMatTrans,
        featureMatTrans,
        numInstances * numFeatures * sizeof( float ),
        hipMemcpyHostToDevice ) );
    cudaErrorCheck( hipMemcpyAsync(
        dWeightArr,
        node.weights,
        (numFeatures + 1) * sizeof( float ),
        hipMemcpyHostToDevice ) );
    cudaErrorCheck( hipMemcpyAsync(
        dClassArr,
        classArr,
        numInstances * sizeof( unsigned short ),
        hipMemcpyHostToDevice ) );

    /*----------- Determine block and grid size of Dot kernel -----------*/
    dim3 dotBlockDim;
    dim3 dotGridDim;
    // Assume numFeatures <= 1024 (max number of threads per block)
    dotBlockDim.x = numFeatures;
    if (numInstances < 1024)
        dotGridDim.x = numInstances;
    else
    {
        dotGridDim.x = 1024;
        dotGridDim.y = (numInstances + dotGridDim.x - 1) / dotGridDim.x;
    }
    // Compute number of warps for shuffle reduction sum
    unsigned int actNumWarps = (numFeatures + WARP_SIZE - 1) / WARP_SIZE;

    /*------- Determine block and grid size of ComputeCost kernel -------*/
    dim3 ccBlockDim;
    dim3 ccGridDim;
    if (numInstances > 1024)
    {
        ccBlockDim.x = 1024;
        ccGridDim.x = (numInstances + 1023) / 1024;
    }
    else ccBlockDim.x = numInstances;

    /*------- Determine block and grid size of UpdateWeight kernel -------*/
    dim3 uwBlockDim;
    dim3 uwGridDim;
    unsigned int uwChunkSize;
    unsigned int uwNumChunks;
    if (numInstances > 512)
    {
        uwNumChunks = 512;
        uwChunkSize = numInstances / uwNumChunks;
    }
    else
    {
        uwNumChunks = numInstances;
        uwChunkSize = 1;
    }
    uwBlockDim.x = uwNumChunks;
    uwGridDim.x = numFeatures;
    // Compute number of warps for shuffle reduction sum
    unsigned int uwNumWarps = (uwNumChunks + WARP_SIZE - 1) / WARP_SIZE;

    unsigned int alpha = 50;
    unsigned int maxIter = 200;
    unsigned int iter = 0;

    time_t start, end;
    float dif;
    time( &start );
    
    printf( "\nStart gradient descent...\n" );

    // Gradient descent
    do
    {
        Dot<<< dotGridDim, dotBlockDim >>>(
            dCostArr,
            dWeightArr,
            dFeatureMat,
            dClassArr,
            actNumWarps,
            numInstances,
            numFeatures );
        cudaErrorCheck( hipGetLastError() );

        ComputeCost<<< ccGridDim, ccBlockDim >>>(
            dCostArr,
            dClassArr,
            numInstances );
        cudaErrorCheck( hipGetLastError() );

        UpdateWeight<<< uwGridDim, uwBlockDim >>>(
            dWeightArr,
            dCostArr,
            dFeatureMatTrans,
            alpha,
            uwChunkSize,
            uwNumWarps,
            numInstances,
            numFeatures );
        cudaErrorCheck( hipGetLastError() );

        iter++;
    }
    while (iter == 1 || iter < maxIter);

    cudaErrorCheck( hipDeviceSynchronize() );
    
    // hipMemcpy(weight);
    // cublasErrorCheck( hipblasDestroy( cublasHandle ) );

    time( &end );
    dif = difftime( end, start );
    printf( "Time taken is %.2lf seconds.\n", dif );

    hipFree( dFeatureMat );
    hipFree( dFeatureMatTrans );
    hipFree( dClassArr );
    hipFree( dWeightArr );
    hipFree( dCostArr );

    free( node.weights );

    return 0;
}
