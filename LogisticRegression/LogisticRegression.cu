#include "hip/hip_runtime.h"

#include "Helper.h"
#include "ArffImporter.h"

#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
// #include "hipblas.h"


Node initNode( unsigned int numFeatures )
{
    Node node;
    node.numFeatures = numFeatures;
    node.weights = (double*) malloc( (numFeatures + 1) * sizeof( double ) );
    memset( node.weights, 1.0, (numFeatures + 1) * sizeof( double ) );
    // printf( "weight: %f\n", node.weights[0] );

    return node;
}

void normalize(
    std::vector<NumericAttr> featureVec,
    double* featureBuff,
    unsigned int numInstances )
{
    unsigned int numFeatures = featureVec.size();

    for (unsigned int i = 0; i < numFeatures; i++)
    {
        // Use either range / standard deviation
        double range = featureVec[i].max - featureVec[i].min;
        if (range == 0.0) continue;

        for (unsigned int j = 0; j < numInstances; j++)
        {
            unsigned int featureIndex = j * numFeatures + i;
            featureBuff[featureIndex] =
                (featureBuff[featureIndex] - featureVec[i].mean) / range;
        }
    }
}

__device__ double activate(
    Node* node,
    double* inputArr )
{
    double linearRes = node->weights[node->numFeatures];
    node->inputs = inputArr;

    unsigned int numFeatures = node->numFeatures;
    for (unsigned int i = 0; i < numFeatures; i++)
        linearRes += node->weights[i] * node->inputs[i];

    node->output = 1.0 / (1.0 + exp(-linearRes));

    return node->output;
}

// __device__ double computeCost( double hRes, unsigned short y )
// {
//     return (y)? -log(hRes) : -log(1.0 - hRes);
//     // return -y * log(hRes) - (1 - y) * (1 - log(hRes));
// }

__device__ void parallelSum(
    double* dProductArr,
    unsigned int elementId,
    unsigned int length )
{
    if (length <= 1024)
        for (unsigned int i = length; i > 1; i /= 2)
        {
            if (elementId < i / 2)
            {
                dProductArr[elementId] +=
                    dProductArr[elementId + i / 2];

                // Odd
                if (i & 1 && elementId == i / 2 - 1)
                    dProductArr[elementId] += dProductArr[i - 1];
            }
            __syncthreads();
        }
    else
    {

    }
}

__global__ void Activate(
    double* dDiffArr,
    double* dWeightArr,
    const double* dFeatureBuff,
    const unsigned short* dClassBuff,
    const unsigned int numInstances,
    const unsigned int numFeatures )
    // const hipblasHandle_t cublasHandle
{
    unsigned int instanceId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int featureId = threadIdx.y * blockDim.x + threadIdx.x;
    if (instanceId >= numInstances || featureId >= numFeatures) return;
    // if (featureId == 0) printf( "Instance ID: %u\n", instanceId );

    extern __shared__ double dProductShared[];
    dProductShared[featureId] =
        dWeightArr[featureId] * dFeatureBuff[instanceId * numFeatures + featureId];
    __syncthreads();

    // Parallel sum
    // Assume numFeatures is big
    parallelSum( dProductShared, featureId, numFeatures );

    double linearRes = 0.0;
    // hipblasDdot( cublasHandle, numFeatures, &dFeatureBuff[instanceId * numFeatures], 1, dWeightArr, 1, &linearRes );
    linearRes += dWeightArr[numFeatures];

    if (featureId > 0) return;

    linearRes += dProductShared[0];
    double hRes = 1.0 / (1.0 + exp(-linearRes));
    dDiffArr[instanceId] = hRes - (double) dClassBuff[instanceId];

    // if (instanceId == 20000)
    //     printf( "Activation completed, hRes: %f\n", dDiffArr[instanceId] );
}

__global__ void UpdateWeight(
    double* dDiffArr,
    double* dWeightArr,
    const double* dFeatureBuff,
    const unsigned int alpha,
    const unsigned int chunkSize,
    const unsigned int numInstances,
    const unsigned int numFeatures )
{
    // One block per feature, one thread per instance
    unsigned int featureId = blockIdx.y * gridDim.x + blockIdx.x;
    unsigned int instChunkId = threadIdx.y * blockDim.x + threadIdx.x;
    if (instChunkId >= numInstances || featureId >= numFeatures) return;

    unsigned int stopId = chunkSize * (instChunkId + 1);
    // Last chunk
    if (instChunkId == blockDim.x - 1)
        stopId = numInstances;

    // An array of values of one feature
    extern __shared__ double dProductShared[];
    // dProductShared[instanceId] =
    //     dFeatureBuff[instanceId * numFeatures + featureId] *
    //         dDiffArr[instanceId];
    dProductShared[instChunkId] = 0.0;
    for (unsigned int i = chunkSize * instChunkId; i < stopId; i++)
        dProductShared[instChunkId] +=
            dFeatureBuff[i * numFeatures + featureId] * dDiffArr[i];
    __syncthreads();

    // Parallel sum
    // Assume numInstances is big
    parallelSum( dProductShared, instChunkId, blockDim.x );

    // Update weights
    if (instChunkId > 0) return;
    dWeightArr[featureId] -=
        alpha / (double) numInstances * dProductShared[0];

    if (featureId == 0)
        printf( "Updating weights completed, weight: %f\n", dWeightArr[0] );
}

// __global__ void SumCost(
//     unsigned short* dClassIndexBuff,
//     const unsigned int numInstances )
// {
//     unsigned int instanceId = threadIdx.y * blockDim.x + threadIdx.x;
//     if (instanceId >= numInstances) return;

//     // Parallel sum
// }

void cudaErrorCheck( hipError_t cudaRes )
{
    if (cudaRes != hipSuccess)
        printf(
            "kernel launch failed with error \"%s\".\n",
            hipGetErrorString( cudaRes )
        );
}

// void cublasErrorCheck( hipblasStatus_t cublasRes )
// {
//     if (cublasRes != HIPBLAS_STATUS_SUCCESS)
//         printf( "Cublas library failed to load.\n" );
// }

int main()
{
    ArffImporter trainSetImporter;
    trainSetImporter.Read( "Dataset/train/train-first1000.arff" );

    // ArffImporter testSetImporter;
    // testSetImporter.Read( "Dataset/test/dev-first1000.arff" );

    unsigned int numInstances = trainSetImporter.GetNumInstances();
    double* featureBuff = trainSetImporter.GetInstances();
    unsigned short* classIndexBuff = trainSetImporter.GetClassIndex();
    std::vector<NumericAttr> featureVec = trainSetImporter.GetFeatures();
    unsigned int numFeatures = featureVec.size();

    // unsigned int numInstances = 25000;
    // unsigned int numFeatures = 1000;
    unsigned int alpha = 50;
    unsigned int maxIter = 200;
    unsigned int iter = 0;
    // double costSumPre = 0.0;
    // double costSumNew;

    // Determine block and grid size
    dim3 actBlockDim;
    dim3 actGridDim;
    dim3 sumCostBlockDim;
    dim3 uwBlockDim;
    dim3 uwGridDim;
    if (numInstances < 1024)
    {
        actGridDim.x = numInstances;
        sumCostBlockDim.x = numInstances;
    }
    else
    {
        actGridDim.x = 1024;
        actGridDim.y = (numInstances + actGridDim.x - 1) / actGridDim.x;
        sumCostBlockDim.x = 1024;
        sumCostBlockDim.y = (numInstances + sumCostBlockDim.x - 1) / sumCostBlockDim.x;
    }

    if (numFeatures < 1024) actBlockDim.x = numFeatures;
    else
    {
        actBlockDim.x = 1024;
        actBlockDim.y = (numFeatures + actBlockDim.x - 1) / actBlockDim.x;
    }

    uwBlockDim.x = actGridDim.x;
    uwGridDim = actBlockDim;
    unsigned int uwChunkSize = numInstances / uwBlockDim.x;

    normalize( featureVec, featureBuff, numInstances );
    Node node = initNode( numFeatures );

    double* dDiffArr;
    double* dWeightArr;
    double* dFeatureBuff;
    unsigned short* dClassBuff;
    cudaErrorCheck( hipMalloc( (void**) &dWeightArr, (numFeatures + 1) * sizeof( double ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dDiffArr, numInstances * sizeof( double ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dFeatureBuff, numInstances * numFeatures * sizeof( double ) ) );
    cudaErrorCheck( hipMalloc( (void**) &dClassBuff, numInstances * sizeof( unsigned short ) ) );

    cudaErrorCheck( hipMemcpy(
        dFeatureBuff,
        featureBuff,
        numInstances * numFeatures * sizeof( double ),
        hipMemcpyHostToDevice ) );
    cudaErrorCheck( hipMemcpy(
        dWeightArr,
        node.weights,
        (numFeatures + 1) * sizeof( double ),
        hipMemcpyHostToDevice ) );
    cudaErrorCheck( hipMemcpy(
        dClassBuff,
        classIndexBuff,
        numInstances * sizeof( unsigned short ),
        hipMemcpyHostToDevice ) );

    // hipblasHandle_t cublasHandle;
    // cublasErrorCheck( hipblasCreate( &cublasHandle ) );

    time_t start, end;
    double dif;
    time( &start );
    
    printf( "\nStart gradient descent...\n" );

    // Gradient descent
    do
    {
        Activate<<< actGridDim, actBlockDim, numFeatures * sizeof( double )  >>>(
            dDiffArr,
            dWeightArr,
            dFeatureBuff,
            dClassBuff,
            numInstances,
            numFeatures );//cublasHandle
        cudaErrorCheck( hipGetLastError() );
        cudaErrorCheck( hipDeviceSynchronize() );

        // SumCost<<< 1, sumCostBlockDim >>>();
        // hipDeviceSynchronize();
        // hipMemcpy(costSumNew);

        UpdateWeight<<< uwGridDim, uwBlockDim, uwBlockDim.x * sizeof( double ) >>>(
            dDiffArr,
            dWeightArr,
            dFeatureBuff,
            alpha,
            uwChunkSize,
            numInstances,
            numFeatures );
        cudaErrorCheck( hipGetLastError() );
        cudaErrorCheck( hipDeviceSynchronize() );

        // memset( batchArr, 0, numFeatures * sizeof( double ) );

        // for (unsigned int i = 0; i < numInstances; i++)
        // {
        //     // double hRes = activate( &node, &featureBuff[i * numFeatures] );

        //     double linearRes = weightBuff[numFeatures];
        //     for (unsigned int j = 0; j < numFeatures; j++)
        //         linearRes += weightBuff[j] * featureBuff[i * numFeatures + j];

        //     double hRes = 1.0 / (1.0 + exp(-linearRes));
        //     costSumNew += computeCost( hRes, classIndexBuff[i] );
        //     diffArr[i] = hRes - (double) classIndexBuff[i];
        //     // double diff = hRes - (double) classIndexBuff[i];
        //     // for (unsigned int j = 0; j < numFeatures; j++)
        //     //     batchArr[j] += diff * featureBuff[i * numFeatures + j];
        // }

        // for (unsigned int j = 0; j < numFeatures; j++)
        // {
        //     batchArr[j] = 0;
        //     for (unsigned int i = 0; i < numInstances; i++)
        //         batchArr[j] += diffArr[i] * featureBuff[i * numFeatures + j];
        //     // Update weights
        //     weightBuff[j] -= alpha / (double) numInstances * batchArr[j];
        // }

        // deltaCostSum = costSumPre - costSumNew;
        // costSumPre = costSumNew;

        // printf( "Delta cost: %f\n", deltaCostSum );
        // printf( "Pre cost: %f\n", costSumPre );
        // printf( "New cost: %f\n", costSumNew );

        // Update weights
        // #pragma acc kernels loop
        // for (unsigned int j = 0; j < numFeatures; j++)
        //     weightBuff[j] -= alpha / (double) numInstances * batchArr[j];

        iter++;
    }
    // while (iter == 1 || (deltaCostSum > 1.0 && iter < maxIter));
    while (iter == 1 || iter < maxIter);

    time( &end );
    dif = difftime( end, start );

    printf( "Time taken is %.2lf seconds.\n", dif );
    
    // hipMemcpy(weight);
    // cublasErrorCheck( hipblasDestroy( cublasHandle ) );
    hipDeviceSynchronize();

    hipFree( dFeatureBuff );
    hipFree( dClassBuff );
    hipFree( dWeightArr );
    hipFree( dDiffArr );

    free( node.weights );

    return 0;
}
